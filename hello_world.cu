#include "hip/hip_runtime.h"
#include "common/book.h"
__global__ void add(int a, int b, int* c)
{
    *c = a + b;
}

int main(void)
{
    int c;
    int* dev_c;
    hipMalloc((void**) &dev_c, sizeof(int));
    add<<<1, 1>>>(2, 3, dev_c);
    hipMemcpy(&c, &dev_c, sizeof(int), hipMemcpyHostToDevice);
    printf("2+3=%d\n", c);
    hipFree(dev_c);
    return(0);
}