#include "common/book.h"

int main( void ) 
{
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("Id of current device: %d\n", dev);
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);
    printf("choosen device id: %d\n", dev);
    hipSetDevice(dev);
}